﻿/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "hitables/sphere.h"
#include "hitables/hitableList.h"
#include "util/camera.h"
#include "materials/material.h"
#include "util/scene.h"
#include "util/renderer.h"
#include "util/window.h"
#include "util/common.h"

void initializeWorldCuda(bool showWindow, bool writeImagePPM, bool writeImagePNG, hitable** world, Window** w, Image** image, Camera** cam, Renderer** render)
{
    // World
/*
    hitable** list2 = new hitable*[4];
    list2[0] = new sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f, new lambertian(vec3(0.5f, 0.5f, 0.5f)));
    list2[1] = new sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new dielectric(1.5f));
    list2[2] = new sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new lambertian(vec3(0.4f, 0.2f, 0.1f)));
    list2[3] = new sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));

    hitable* camera =  new hitableList(list2, 4);
*/

    int numHitables = 4;
    hitable** list;
    checkCudaErrors(hipMallocManaged(&list, numHitables*sizeof(hitable*)));
    checkCudaErrors(hipMallocManaged(world, sizeof(hitable)));
    for (int i = 0; i < numHitables; i++)
    {
        checkCudaErrors(hipMallocManaged(&list[i], sizeof(hitable)));
        new (list[i]) sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f, new lambertian(vec3(0.5f, 0.5f, 0.5f)));
    }

    //new (list[0]) sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f, new lambertian(vec3(0.5f, 0.5f, 0.5f)));
    //new (list[1]) sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new dielectric(1.5f));
    //new (list[2]) sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new lambertian(vec3(0.4f, 0.2f, 0.1f)));
    //new (list[3]) sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));
    new (*world) hitableList(list, numHitables);

/*
    hitable** list;
    int num_hitables = 4;
    checkCudaErrors(hipMallocManaged((void **)&list, num_hitables*sizeof(hitable*)));
    checkCudaErrors(hipMallocManaged((void **)&world, sizeof(hitable*)));
    simpleScene<<<1,1>>>(list, world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
*/

    // Camera
    vec3 lookFrom(13.0f, 2.0f, 3.0f);
    vec3 lookAt(0.0f, 0.0f, 0.0f);
    checkCudaErrors(hipMallocManaged(cam, sizeof(Camera)));
    new (*cam) Camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f), 20.0f, float(nx)/float(ny), distToFocus);

    // Renderer
    checkCudaErrors(hipMallocManaged(render, sizeof(Renderer)));
    new (*render) Renderer(showWindow, writeImagePPM, writeImagePNG);

    // Image
    checkCudaErrors(hipMallocManaged(image, sizeof(Image)));
    new (*image) Image(showWindow, writeImagePPM || writeImagePNG, nx, ny, tx, ty);

    // Window
    if (showWindow)
        *w = new Window(*cam, *render, nx, ny, thetaInit, phiInit, zoomScale, stepScale);
}

void destroyWorldCuda(bool showWindow, hitable* world, Window* w, Image* image, Camera* cam, Renderer* render)
{
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(render));
    checkCudaErrors(hipFree(image));
}
