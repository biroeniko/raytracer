﻿/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "hitables/sphere.h"
#include "hitables/hitableList.h"
#include "util/camera.h"
#include "materials/material.h"
#include "util/scene.h"
#include "util/renderer.h"
#include "util/window.h"
#include "util/common.h"

void initializeWorldCuda(bool showWindow, bool writeImagePPM, bool writeImagePNG, hitable** world, Window** w, Image** image, Camera** cam, Renderer** render)
{
    hitable** list;
    int num_hitables = 500;
    checkCudaErrors(hipMallocManaged((void **)&list, num_hitables*sizeof(hitable *)));
    checkCudaErrors(hipMallocManaged((void **)&world, sizeof(hitable *)));
    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(Camera *)));
    simpleScene<<<1,1>>>(list, world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

void destroyWorldCuda(bool showWindow, hitable* world, Window* w, Image* image, Camera* cam, Renderer* render)
{

}
