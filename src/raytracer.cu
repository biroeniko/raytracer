﻿/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "hitables/sphere.h"
#include "hitables/hitableList.h"
#include "util/camera.h"
#include "materials/material.h"
#include "util/scene.h"
#include "util/renderer.h"
#include "util/window.h"
#include "util/common.h"

void initializeWorldCuda(bool showWindow, bool writeImagePPM, bool writeImagePNG, hitable** world, Window** w, Image** image, Camera** cam, Renderer** render)
{
    hitable** list;
    int num_hitables = 4;
    checkCudaErrors(hipMallocManaged((void **)&list, num_hitables*sizeof(hitable *)));
    checkCudaErrors(hipMallocManaged((void **)&world, sizeof(hitable *)));
    simpleScene<<<1,1>>>(list, world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(Camera *)));
    checkCudaErrors(hipMallocManaged((void **)&image, sizeof(Image *)));
    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(Renderer *)));
    *image = new Image(showWindow, writeImagePPM || writeImagePNG, nx, ny, tx, ty);

    vec3 lookFrom(13.0f, 2.0f, 3.0f);
    vec3 lookAt(0.0f, 0.0f, 0.0f);
    float distToFocus = 10.0f;
    float aperture = 0.1f;

    *cam = new Camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f), 20.0f, float(nx)/float(ny), distToFocus);
    *render = new Renderer(showWindow, writeImagePPM, writeImagePNG);

    *world = simpleScene2();

    if (showWindow)
    {
        checkCudaErrors(hipMallocManaged((void **)&w, sizeof(Window *)));
        *w = new Window(*cam, *render, nx, ny, thetaInit, phiInit, zoomScale, stepScale);
    }

}

void destroyWorldCuda(bool showWindow, hitable* world, Window* w, Image* image, Camera* cam, Renderer* render)
{

}
