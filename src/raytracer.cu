﻿/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "hitables/sphere.h"
#include "hitables/hitableList.h"
#include "util/camera.h"
#include "materials/material.h"
#include "util/scene.h"
#include "util/renderer.h"
#include "util/window.h"
#include "util/common.h"

CUDA_GLOBAL void createWorld(bool showWindow, bool writeImagePPM, bool writeImagePNG, Camera** cam)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *cam = new Camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f), 20.0f, float(nx)/float(ny), distToFocus);
    }
}

void initializeWorldCuda(bool showWindow, bool writeImagePPM, bool writeImagePNG, hitable** world, Window** w, Image** image, Camera** cam, Renderer** render)
{
    hitable** list;
    int num_hitables = 4;
    checkCudaErrors(hipMallocManaged((void **)&list, num_hitables*sizeof(hitable *)));
    checkCudaErrors(hipMallocManaged((void **)&world, sizeof(hitable *)));
    simpleScene<<<1,1>>>(list, world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(Camera *)));
    checkCudaErrors(hipMallocManaged((void **)&image, sizeof(Image *)));
    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(Renderer *)));

    if (showWindow)
        checkCudaErrors(hipMallocManaged((void **)&w, sizeof(Window *)));

    createWorld<<<1,1>>>(showWindow, writeImagePPM, writeImagePNG, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

void destroyWorldCuda(bool showWindow, hitable* world, Window* w, Image* image, Camera* cam, Renderer* render)
{

}
