﻿/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "hitables/sphere.h"
#include "hitables/hitableList.h"
#include "util/camera.h"
#include "materials/material.h"
#include "util/scene.h"
#include "util/renderer.h"
#include "util/window.h"
#include "util/common.h"

CUDA_GLOBAL void createWorld(Camera** cam)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        vec3 lookFrom(13.0f, 2.0f, 3.0f);
        vec3 lookAt(0.0f, 0.0f, 0.0f);
        //*cam = new Camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f), 20.0f, float(nx)/float(ny), distToFocus);
    }
}

void initializeWorldCuda(bool showWindow, bool writeImagePPM, bool writeImagePNG, hitable** world, Window** w, Image** image, Camera** cam, Renderer** render)
{
    hitable** list;
    int num_hitables = 4;
    checkCudaErrors(hipMallocManaged((void **)&list, num_hitables*sizeof(hitable *)));
    checkCudaErrors(hipMallocManaged((void **)&world, sizeof(hitable *)));
    simpleScene<<<1,1>>>(list, world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(Camera *)));
    createWorld<<<1,1>>>(cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    *image = new Image(showWindow, writeImagePPM || writeImagePNG, nx, ny, tx, ty);
    *render = new Renderer(showWindow, writeImagePPM, writeImagePNG);

    if (showWindow)
        *w = new Window(*cam, *render, nx, ny, thetaInit, phiInit, zoomScale, stepScale);

}

void destroyWorldCuda(bool showWindow, hitable* world, Window* w, Image* image, Camera* cam, Renderer* render)
{

}
