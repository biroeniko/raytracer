/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "util/image.h"

#ifdef CUDA_ENABLED

    CUDA_GLOBAL void cudaResetImageKernel(Vec3 *pixels, int nx, int ny)
    {

        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        if ((i >= nx) || (j >= ny))
            return;
        int pixelIndex = j*nx + i;
        pixels[pixelIndex] = Vec3(0.0f, 0.0f, 0.0f);

    }

    void Image::cudaResetImage()
    {

        dim3 blocks(nx/tx+1, ny/ty+1);
        dim3 threads(tx,ty);
        cudaResetImageKernel<<<blocks, threads>>>(pixels, nx, ny);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

    }

#endif // CUDA_ENABLED
