#include "hip/hip_runtime.h"
/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "util/common.h"
#include "util/renderer.h"

#ifdef CUDA_ENABLED
    CUDA_GLOBAL void render(vec3* frameBuffer, int nx, int ny)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        if ((i >= nx) || (j >= ny))
            return;
        int pixelIndex = j*nx + i;

        vec3 col(0.0f,1.0f,0.0f);

        frameBuffer[pixelIndex] = col;

        /*
        RandomGenerator rng(sampleCount, i*image->rows + j);
        float u = float(i + rng.get1f()) / float(image->rows); // left to right
        float v = float(j + rng.get1f()) / float(image->columns); // bottom to top

        ray r = cam->getRay(rng, u,v);

        image->pixels[i][j] += color(rng, r, world, 0);

        vec3 col = image->pixels[i][j] / sampleCount;

        // Gamma encoding of images is used to optimize the usage of bits
        // when encoding an image, or bandwidth used to transport an image,
        // by taking advantage of the non-linear manner in which humans perceive
        // light and color. (wikipedia)

        // we use gamma 2: raising the color to the power 1/gamma (1/2)
        col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));

        int ir = int(255.99f*col[0]);
        int ig = int(255.99f*col[1]);
        int ib = int(255.99f*col[2]);

        if (writeImagePNG)
        {
            // PNG
            int index = (image->columns - 1 - j) * image->rows + i;
            int index3 = 3 * index;

            fileOutputImage[index3 + 0] = ir;
            fileOutputImage[index3 + 1] = ig;
            fileOutputImage[index3 + 2] = ib;
        }

        if (showWindow)
            windowPixels[(image->columns-j-1)*image->rows + i] = (ir << 16) | (ig << 8) | (ib);

        */
    }
#endif // CUDA_ENABLED

#ifdef CUDA_ENABLED
    void Renderer::cudaRender(uint32_t* windowPixels, Camera* cam, hitable* world, Image* image, int sampleCount, uint8_t *fileOutputImage)
    {
        std::cout << image->nx << std::endl;
        std::cout << image->ny << std::endl;

        dim3 blocks(image->nx/image->tx+1, image->ny/image->ty+1);
        dim3 threads(image->tx, image->ty);

        render<<<blocks, threads>>>(image->frameBuffer, image->nx, image->ny);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        std::cout << image->frameBuffer[0] << std::endl;
    }
#endif // CUDA_ENABLED
