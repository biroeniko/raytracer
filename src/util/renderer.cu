#include "hip/hip_runtime.h"
﻿/* MIT License
Copyright (c) 2018 Biro Eniko
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "util/common.h"
#include "util/globals.h"
#include "util/renderer.h"

#include "hitables/sphere.h"
#include "hitables/hitableList.h"
#include "util/camera.h"
#include "materials/material.h"
#include "util/scene.cuh"
#include "util/window.h"

CUDA_DEV int numHitables = 0;

#ifdef CUDA_ENABLED
    void initializeWorldCuda(bool showWindow, bool writeImagePPM, bool writeImagePNG, hitable*** list, hitable** world, Window** w, Image** image, Camera** cam, Renderer** renderer)
    {
        int choice = 4;

        switch(choice)
        {
            case 0:
                numHitables = 4;
                break;
            case 1:
                numHitables = 58;
                break;
            case 2:
                numHitables = 901;
                break;
            case 3:
                numHitables = 102;
                break;
            case 4:
                numHitables = 68;
                break;
        }

        // World
        checkCudaErrors(hipMallocManaged(list, numHitables*sizeof(hitable*)));
        hitable** worldPtr;
        checkCudaErrors(hipMallocManaged(&worldPtr, sizeof(hitable*)));
        switch(choice)
        {
            case 0:
                simpleScene<<<1,1>>>(*list, worldPtr);
                break;
            case 1:
                simpleScene2<<<1,1>>>(*list, worldPtr);
                break;
            case 2:
                randomScene<<<1,1>>>(*list, worldPtr);
                break;
            case 3:
                randomScene2<<<1,1>>>(*list, worldPtr);
                break;
            case 4:
                randomScene3<<<1,1>>>(*list, worldPtr);
                break;
        }
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        *world = *worldPtr;
        checkCudaErrors(hipFree(worldPtr));

        // Camera
        checkCudaErrors(hipMallocManaged(cam, sizeof(Camera)));
        new (*cam) Camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f), 20.0f, float(nx)/float(ny), distToFocus);

        // Renderer
        checkCudaErrors(hipMallocManaged(renderer, sizeof(Renderer)));
        new (*renderer) Renderer(showWindow, writeImagePPM, writeImagePNG);

        // Image
        checkCudaErrors(hipMallocManaged(image, sizeof(Image)));
        new (*image) Image(showWindow, writeImagePPM || writeImagePNG, nx, ny, tx, ty);

        // Window
        if (showWindow)
            *w = new Window(*cam, *renderer, nx, ny, thetaInit, phiInit, zoomScale, stepScale);
    }

    CUDA_GLOBAL void freeWorldCuda(hitable** list, hitable** world)
    {
        if (threadIdx.x == 0 && blockIdx.x == 0)
        {
            for (int i = 0; i < numHitables; i++)
            {
                delete ((sphere *)list[i])->matPtr;
                delete list[i];
            }
            //delete *world;
        }
    }

    void destroyWorldCuda(bool showWindow, hitable** list, hitable* world, Window* w, Image* image, Camera* cam, Renderer* render)
    {
        freeWorldCuda<<<1,1>>>(list, &world);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipFree(cam));
        checkCudaErrors(hipFree(render));
        checkCudaErrors(hipFree(image));
    }

    CUDA_GLOBAL void render(Camera* cam, Image* image, hitable* world, Renderer* render, int sampleCount)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;

        if ((i >= image->nx) || (j >= image->ny))
            return;

        int pixelIndex = j*image->nx + i;

        // Render the samples in batches
        for (int s = 0; s < nsBatch; s++)
        {
            RandomGenerator rng(sampleCount * nsBatch + s, pixelIndex);
            float u = float(i + rng.get1f()) / float(image->nx); // left to right
            float v = float(j + rng.get1f()) / float(image->ny); // bottom to top
            ray r = cam->getRay(rng, u, v);

            image->pixels[pixelIndex] += render->color(rng, r, world, 0);
        }

        vec3 col = image->pixels[pixelIndex] / (sampleCount * nsBatch);

        image->pixels2[pixelIndex] = col;
    }

    CUDA_GLOBAL void display(Image* image)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;

        int pixelIndex = j*image->nx + i;

        vec3 col = image->pixels2[pixelIndex];

        // Gamma encoding of images is used to optimize the usage of bits
        // when encoding an image, or bandwidth used to transport an image,
        // by taking advantage of the non-linear manner in which humans perceive
        // light and color. (wikipedia)

        // we use gamma 2: raising the color to the power 1/gamma (1/2)
        col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));

        int ir = clamp(int(255.f*col[0]), 0, 255);
        int ig = clamp(int(255.f*col[1]), 0, 255);
        int ib = clamp(int(255.f*col[2]), 0, 255);

        if (image->writeImage)
        {
            // PNG
            int index = (image->ny - 1 - j) * image->nx + i;
            int index3 = 3 * index;

            image->fileOutputImage[index3 + 0] = ir;
            image->fileOutputImage[index3 + 1] = ig;
            image->fileOutputImage[index3 + 2] = ib;
        }

        if (image->showWindow)
            image->windowPixels[(image->ny-j-1)*image->nx + i] = (ir << 16) | (ig << 8) | (ib);
    }

#endif // CUDA_ENABLED

#ifdef CUDA_ENABLED
    void Renderer::cudaRender(Camera* cam, hitable* world, Image* image, int sampleCount)
    {
        dim3 blocks( (image->nx + image->tx - 1)/image->tx, (image->ny + image->ty - 1)/image->ty);
        dim3 threads(image->tx, image->ty);

        // Kernel call for the computation of pixel colors.
        render<<<blocks, threads>>>(cam, image, world, this, sampleCount);

        // Denoise here.
        #ifdef OIDN_ENABLED
            checkCudaErrors(hipDeviceSynchronize());
            image->denoise();
            checkCudaErrors(hipDeviceSynchronize());
        #endif // OIDN_ENABLED

        // Kernel call to fill the output buffers.
        display<<<blocks, threads>>>(image);

        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

    }
#endif // CUDA_ENABLED
